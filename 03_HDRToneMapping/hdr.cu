#include "hip/hip_runtime.h"
%%cuda --name student_func.cu

/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.


  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

#include "utils.h"

#include <stdio.h>
#include <float.h>
#include <limits.h>

__device__ float _min(float a, float b) {
	return a < b ? a : b;
}

__device__ float _max(float a, float b) {
	return a > b ? a : b;
}

__global__
void findMinMaxLogLumPerBlock(const float* const d_logLuminance,
		const size_t numRows, const size_t numCols,
		float* d_minLogLum, float* d_maxLogLum)
{
  unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
  unsigned int j = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (i >= numCols || j >= numRows)
    return;

  unsigned int g_oneDOffset = j * numCols + i;
  unsigned int s_oneDOffset = threadIdx.y * blockDim.x + threadIdx.x;
  const unsigned int threadsPerBlock = blockDim.x * blockDim.y;

  extern __shared__ float s_minMaxLogLum[];

  s_minMaxLogLum[s_oneDOffset] = d_logLuminance[g_oneDOffset];
  s_minMaxLogLum[threadsPerBlock + s_oneDOffset] = d_logLuminance[g_oneDOffset];
  __syncthreads();

  for (size_t it = threadsPerBlock / 2; it > 0; it >>= 1)
  {
    if (s_oneDOffset < it)
      s_minMaxLogLum[s_oneDOffset] = min(s_minMaxLogLum[s_oneDOffset],
                                             s_minMaxLogLum[s_oneDOffset + it]);
    __syncthreads();
  }

  if(s_oneDOffset == 0)
    d_minLogLum[blockIdx.y * gridDim.x + blockIdx.x] = s_minMaxLogLum[0];
  __syncthreads();

  for (size_t it = threadsPerBlock / 2; it > 0; it >>= 1)
  {
    if (s_oneDOffset < it)
      s_minMaxLogLum[threadsPerBlock + s_oneDOffset] =
                           max(s_minMaxLogLum[threadsPerBlock + s_oneDOffset],
                           s_minMaxLogLum[threadsPerBlock + s_oneDOffset + it]);
    __syncthreads();
  }

  if(s_oneDOffset == 0)
    d_maxLogLum[blockIdx.y * gridDim.x + blockIdx.x] =
                                                s_minMaxLogLum[threadsPerBlock];
}

__global__
void reduceMinMaxLumPerBlock(float* const d_minLogLumArray,
                             float* const d_maxLogLumArray,
			     const size_t numRows,
			     const size_t numCols,
                             float* d_minLogLum,
                             float* d_maxLogLum)
{
  unsigned int i = threadIdx.x;

  if (i >= (numCols * numRows))
    return;

  const unsigned int blocksPerGrid = numRows * numCols;

  extern __shared__ float s_minMaxLogLumArray[];

  s_minMaxLogLumArray[i] = d_minLogLumArray[i];
  s_minMaxLogLumArray[i + blocksPerGrid] = d_maxLogLumArray[i];
  __syncthreads();

  for (size_t it = blocksPerGrid / 2; it > 0; it >>= 1)
  {
    if (i < it)
      s_minMaxLogLumArray[i] = min(s_minMaxLogLumArray[i],
                                                   s_minMaxLogLumArray[i + it]);
    __syncthreads();
  }

  if(i == 0)
    *d_minLogLum = s_minMaxLogLumArray[0];

  __syncthreads();

  for (size_t it = blocksPerGrid / 2; it > 0; it >>= 1)
  {
    if (i < it)
      s_minMaxLogLumArray[i + blocksPerGrid] =
                                   max(s_minMaxLogLumArray[i + blocksPerGrid],
                                   s_minMaxLogLumArray[i + blocksPerGrid + it]);
    __syncthreads();
  }

  if(i == 0)
    *d_maxLogLum = s_minMaxLogLumArray[blocksPerGrid];

}

__global__
void calculateHisto(const float* const d_logLuminance,
                    const size_t numRows,
                    const size_t numCols,
                    const size_t numBins,
                    float* d_minLogLum,
                    float* d_rangeLogLum,
                    unsigned int* d_histo)
{
  unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
  unsigned int j = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (i >= numCols || j >= numRows)
    return;

  unsigned int g_oneDOffset = j * numCols + i;

  unsigned int binNum = min(static_cast<unsigned int>(numBins - 1),
                        static_cast<unsigned int>(((d_logLuminance[g_oneDOffset]
                              - (*d_minLogLum)) / (*d_rangeLogLum)) * numBins));

  atomicAdd(&(d_histo[binNum]), 1);

}

__global__
void hellisAndSteeleCDF(unsigned int* d_histo, const size_t numBins,
                        unsigned int* d_cdf)
{
  extern __shared__ unsigned int temp[];
	unsigned int g_oneDOffset = (blockIdx.x * blockDim.x) + threadIdx.x;
  
  if (g_oneDOffset >= numBins)
    return;
	
  unsigned int pout = 0,pin=1;
	
  if(g_oneDOffset != 0)
    temp[g_oneDOffset] = d_histo[g_oneDOffset-1]; //exclusive scan
	else
    temp[g_oneDOffset] = 0;
  
  __syncthreads();

	for (size_t off = 1; off < numBins; off <<= 1) {
		pout = 1 - pout;
		pin = 1 - pout;
		if (g_oneDOffset >= off)
      temp[numBins * pout + g_oneDOffset] = temp[numBins * pin + g_oneDOffset]
                                     + temp[numBins * pin + g_oneDOffset - off];
		else
      temp[numBins * pout + g_oneDOffset] = temp[numBins * pin + g_oneDOffset];
		__syncthreads();
	}
	d_cdf[g_oneDOffset] = temp[pout * numBins + g_oneDOffset];

}

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */

  // Calculate min and max logLum per block and copy it back to global memory
  float* d_minLogLumPtr = nullptr;
  float* d_maxLogLumPtr = nullptr;

  // Number of threads per block (32 * 32)
  const unsigned int threads = 32;

  // Number of blocks per grid
  unsigned int blocksX = (numCols + threads - 1) / threads;
  unsigned int blocksY = (numRows + threads - 1) / threads;

  // Allocate memory for min and max logLum
  checkCudaErrors(hipMalloc(&d_minLogLumPtr,
                                            sizeof(float) * blocksX * blocksY));
  checkCudaErrors(hipMalloc(&d_maxLogLumPtr,
                                            sizeof(float) * blocksX * blocksY));
  checkCudaErrors(hipMemset(d_minLogLumPtr, 0,
                                            sizeof(float) * blocksX * blocksY));
  checkCudaErrors(hipMemset(d_maxLogLumPtr, 0,
                                            sizeof(float) * blocksX * blocksY));

  dim3 threadsPerBlock(threads, threads, 1);
  dim3 blocksPerGrid(blocksX, blocksY, 1);

  const unsigned int numThreadsPerBlock = threads * threads;
  findMinMaxLogLumPerBlock<<<blocksPerGrid, threadsPerBlock, 2 *
                           numThreadsPerBlock * sizeof(float)>>>(d_logLuminance,
                              numRows, numCols, d_minLogLumPtr, d_maxLogLumPtr);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  float* d_minLogLum = nullptr;
  float* d_maxLogLum = nullptr;
  checkCudaErrors(hipMalloc(&d_minLogLum, sizeof(float)));
  checkCudaErrors(hipMalloc(&d_maxLogLum, sizeof(float)));
  checkCudaErrors(hipMemset(d_minLogLum, 0, sizeof(float)));
  checkCudaErrors(hipMemset(d_maxLogLum, 0, sizeof(float)));

  const unsigned int numblocksPerGrid = blocksY * blocksX;
  reduceMinMaxLumPerBlock<<<1, blocksX * blocksY, 2 * numblocksPerGrid *
                               sizeof(float)>>>(d_minLogLumPtr, d_maxLogLumPtr,
                                    blocksY, blocksX, d_minLogLum, d_maxLogLum);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  checkCudaErrors(hipMemcpy(&min_logLum, d_minLogLum, sizeof(float),
                                                       hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(&max_logLum, d_maxLogLum, sizeof(float),
                                                       hipMemcpyDeviceToHost));
  
  float range_logLum = max_logLum - min_logLum;
  float* d_rangeLogLum = nullptr;
  unsigned int* d_histo = nullptr;
  checkCudaErrors(hipMalloc(&d_rangeLogLum, sizeof(float)));
  checkCudaErrors(hipMalloc(&d_histo, numBins * sizeof(unsigned int)));
  checkCudaErrors(hipMemcpy(d_rangeLogLum, &range_logLum, sizeof(float),
                                                       hipMemcpyHostToDevice));
  checkCudaErrors(hipMemset(d_histo, 0, numBins * sizeof(unsigned int)));
  calculateHisto<<<blocksPerGrid, threadsPerBlock>>>(d_logLuminance, numRows,
                        numCols, numBins, d_minLogLum, d_rangeLogLum, d_histo);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  checkCudaErrors(hipMemset(d_cdf, 0, numBins * sizeof(unsigned int)));
  unsigned int threadsPerBlockCDF = threads * threads;
  unsigned int blocksPerGridCDF = (numBins + ((threads * threads) - 1)) / 
                                                            (threads * threads);
  hellisAndSteeleCDF<<<blocksPerGridCDF, threadsPerBlockCDF, 2 * numBins *
                               sizeof(unsigned int)>>>(d_histo, numBins, d_cdf);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  // Free allocated memory
  checkCudaErrors(hipFree(d_minLogLumPtr));
  checkCudaErrors(hipFree(d_maxLogLumPtr));
  checkCudaErrors(hipFree(d_minLogLum));
  checkCudaErrors(hipFree(d_maxLogLum));
  checkCudaErrors(hipFree(d_rangeLogLum));
  checkCudaErrors(hipFree(d_histo));
}
